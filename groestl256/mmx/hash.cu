#include "hip/hip_runtime.h"
/* hash.c   January 2011
 *
 * Groestl implementation using MMX intrinsics.
 * Has been tested on Intel processors using the gcc compiler only
 * (include the '-std=c99' and '-mmmx' flags when compiling in gcc on a
 * 32-bit machine).
 * 
 * Author: Soeren S. Thomsen
 *
 * This code is placed in the public domain
 */
#define  __MMX__ 81
#if defined (__MMX__)
#include <stdio.h>
#include <stdlib.h>
//#include <mmintrin.h>
#include "hash.h"
#include "tables.h"
#include <string.h>
#include <pthread.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>

// void OutputTransformation512(u32 *outputTransformation, u64* T_shared);

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
// #define getLastCudaError(msg) __getLastCudaError(msg, __FILE__, __LINE__)

void check(hipError_t err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}


#define getLastCudaError() checkLast(__FILE__, __LINE__)



#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}


__device__ void ROUNDP512_function(u64* m_in, u64* m, int r, u64* T_shared, int shared_message_index_64   ){

    u64* T_m64 = (u64*)T_shared;						
    u64 zero;	
    int start_message_index = shared_message_index_64 / COLS512;  //need to find the start offset for the current message block
    u32* x = (u32*)&m_in[start_message_index];
    zero = 0;						
    int local_message_block_index_32 = shared_message_index_64 % COLS512; 
    
    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 0],3)] ^zero;		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 1],1)] ^ zero;		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[7*256+EXT_BYTE(x[ 1],3)] ^ zero;		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 0],1)] ^ zero;		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 0],2)] ^ zero;		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 1],0)] ^ zero;		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 1],2)] ^ zero;		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[0*256+((r)^EXT_BYTE(x[ 0],0))] ^ zero; 
        break;
   }
   __syncthreads();
    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[14],2)] ^m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[15],0)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[15],2)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[0*256+((r)^0x70^EXT_BYTE(x[14],0))] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[14],1)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[14],3)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[15],1)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[7*256+EXT_BYTE(x[15],3)] ^ m[shared_message_index_64]; 
        break;
   }

   __syncthreads();
    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[12],1)] ^m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[12],3)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[13],1)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[7*256+EXT_BYTE(x[13],3)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[0*256+((r)^0x60^EXT_BYTE(x[12],0))] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[12],2)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[13],0)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[13],2)] ^ m[shared_message_index_64]; 
        break;
   }

   __syncthreads();
    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[0*256+((r)^0x50^EXT_BYTE(x[10],0))] ^m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[10],2)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[11],0)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[11],2)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[7*256+EXT_BYTE(x[11],3)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[10],1)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[10],3)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[11],1)] ^ m[shared_message_index_64]; 
        break;
   }

   __syncthreads();
    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[7*256+EXT_BYTE(x[ 9],3)] ^m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 8],1)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 8],3)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 9],1)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 9],2)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[0*256+((r)^0x40^EXT_BYTE(x[ 8],0))] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 8],2)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 9],0)] ^ m[shared_message_index_64]; 
        break;
   }
    
   __syncthreads();
    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 7],2)] ^m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[0*256+((r)^0x30^EXT_BYTE(x[ 6],0))] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 6],2)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 7],0)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 7],1)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[7*256+EXT_BYTE(x[ 7],3)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 6],1)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 6],3)] ^ m[shared_message_index_64]; 
        break;
   }

   __syncthreads();
    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 5],1)] ^m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[7*256+EXT_BYTE(x[ 5],3)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 4],1)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 4],3)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 5],0)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 5],2)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[0*256+((r)^0x20^EXT_BYTE(x[ 4],0))] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 4],2)] ^ m[shared_message_index_64]; 
        break;
   }

   __syncthreads();
    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 3],0)] ^m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 3],2)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[0*256+((r)^0x10^EXT_BYTE(x[ 2],0))] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 2],2)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 2],3)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 3],1)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[7*256+EXT_BYTE(x[ 3],3)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 2],1)] ^ m[shared_message_index_64]; 
        break;
   }

   __syncthreads();

}


/* compute a round of P512 */
// #define ROUNDP512(m_in, m, r, T_shared) do {					\
//     u64* T_m64 = (u64*)T_shared;						\
//     u64 zero;								\
//     u32* x = (u32*)m_in;						\
//     zero = 0;						\
//   } while (0)

/* compute a round of Q512 */

__device__ void ROUNDQ512_function(u64* m_in, u64* m, int r, u64* T_shared, int shared_message_index_64   ){

    u64* T_m64 = (u64*)T_shared;						
    u64 zero;								
    u64 ff;						
    int start_message_index = shared_message_index_64 / COLS512;  //need to find the start offset for the current message block
    u32* x = (u32*)&m_in[start_message_index];
    zero = 0;						
    ff   = -1;				
    int local_message_block_index_32 = shared_message_index_64 % COLS512; 
									
    m_in[shared_message_index_64] = (m_in[shared_message_index_64]^ff);		// even is for shared_message_index_64 64 bits array, we can use it in the 2 bit array in this case
    // m_in[1] = (m_in[1]^ff);					
    // m_in[2] = (m_in[2]^ff);					
    // m_in[3] = (m_in[3]^ff);					
    // m_in[4] = (m_in[4]^ff);					
    // m_in[5] = (m_in[5]^ff);					
    // m_in[6] = (m_in[6]^ff);					
    // m_in[7] = (m_in[7]^ff);					
    if (local_message_block_index_32 == 7) {
      m[shared_message_index_64] = T_m64[0*256+EXT_BYTE(x[ 0],0)] ^ zero;	
    }
    else if (local_message_block_index_32 == 5) {
      m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 0],1)] ^ zero;		
    }
    else if (local_message_block_index_32 == 3) {
      m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 0],2)] ^ zero;		
    }
    else if (local_message_block_index_32 == 1) {
      m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 0],3)] ^ zero;		
    }
    else if (local_message_block_index_32 == 0) {
      m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 1],0)] ^ zero;		
    }
    else if (local_message_block_index_32 == 6) {
      m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 1],1)] ^ zero;		
    }
    else if (local_message_block_index_32 == 4) {
      m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 1],2)] ^ zero;		
    }
    else if (local_message_block_index_32 == 2) {
      m[shared_message_index_64] = T_m64[7*256+((r)^EXT_BYTE(x[ 1],3))] ^ zero;	
    }

    __syncthreads();

    if (local_message_block_index_32 == 7) {
      m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[15],0)] ^ m[shared_message_index_64];		
    }
    else if (local_message_block_index_32 == 5) {
      m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[15],1)] ^ m[shared_message_index_64];		
    }
    else if (local_message_block_index_32 == 3) {
      m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[15],2)] ^ m[shared_message_index_64];		
    }
    else if (local_message_block_index_32 == 1) {
       m[shared_message_index_64] = T_m64[7*256+((r)^0x70^EXT_BYTE(x[15],3))] ^ m[shared_message_index_64]; 
    }
    else if (local_message_block_index_32 == 0) {
      m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[14],3)] ^ m[shared_message_index_64];		
    }
    else if (local_message_block_index_32 == 6) {
      m[shared_message_index_64] = T_m64[0*256+EXT_BYTE(x[14],0)] ^ m[shared_message_index_64];		
    }
    else if (local_message_block_index_32 == 4) {
      m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[14],1)] ^ m[shared_message_index_64];		
    }
    else if (local_message_block_index_32 == 2) {
      m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[14],2)] ^ m[shared_message_index_64];		
    }

    __syncthreads();

   switch	(local_message_block_index_32){
      case 5 :
        m[shared_message_index_64] = T_m64[0*256+EXT_BYTE(x[12],0)] ^ m[shared_message_index_64];		
        break;
    case 3:
      m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[12],1)] ^ m[shared_message_index_64];		
      break;
    case 1:
      m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[12],2)] ^ m[shared_message_index_64];		
      break;
    case 7:
      m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[12],3)] ^ m[shared_message_index_64];		
      break;
    case 6:
      m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[13],0)] ^ m[shared_message_index_64];		
      break;
    case 4:
      m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[13],1)] ^ m[shared_message_index_64];		
      break;
    case 2:
    m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[13],2)] ^ m[shared_message_index_64];		
      break;
    case 0:
      m[shared_message_index_64] = T_m64[7*256+((r)^0x60^EXT_BYTE(x[13],3))] ^ m[shared_message_index_64]; 
      break;

   }
    __syncthreads();

    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[11],0)] ^ m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[11],1)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[11],2)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[7*256+((r)^0x50^EXT_BYTE(x[11],3))] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[10],3)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[0*256+EXT_BYTE(x[10],0)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[10],1)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[10],2)] ^ m[shared_message_index_64]; 
        break;
   }

									
    __syncthreads();

    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 8],3)] ^ m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[0*256+EXT_BYTE(x[ 8],0)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 8],1)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 8],2)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[7*256+((r)^0x40^EXT_BYTE(x[ 9],3))] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 9],0)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 9],1)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 9],2)] ^ m[shared_message_index_64]; 
        break;
   }
									
																	
    __syncthreads();

    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[7*256+((r)^0x30^EXT_BYTE(x[ 7],3))] ^ m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 7],0)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 7],1)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 7],2)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 6],2)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 6],3)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[0*256+EXT_BYTE(x[ 6],0)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 6],1)] ^ m[shared_message_index_64]; 
        break;
   }
									
    __syncthreads();

    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 4],2)] ^ m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 4],3)] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[0*256+EXT_BYTE(x[ 4],0)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 4],1)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 5],2)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[7*256+((r)^0x20^EXT_BYTE(x[ 5],3))] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 5],0)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 5],1)] ^ m[shared_message_index_64]; 
        break;
   }
									
    __syncthreads();

    switch	(local_message_block_index_32){
      case 5 :
          m[shared_message_index_64] = T_m64[6*256+EXT_BYTE(x[ 3],2)] ^ m[shared_message_index_64];		
          break;
      case 3:
        m[shared_message_index_64] = T_m64[7*256+((r)^0x10^EXT_BYTE(x[ 3],3))] ^ m[shared_message_index_64];		
        break;
      case 1:
        m[shared_message_index_64] = T_m64[4*256+EXT_BYTE(x[ 3],0)] ^ m[shared_message_index_64];		
        break;
      case 7:
        m[shared_message_index_64] = T_m64[5*256+EXT_BYTE(x[ 3],1)] ^ m[shared_message_index_64];		
        break;
      case 6:
        m[shared_message_index_64] = T_m64[1*256+EXT_BYTE(x[ 2],1)] ^ m[shared_message_index_64];		
        break;
      case 4:
        m[shared_message_index_64] = T_m64[2*256+EXT_BYTE(x[ 2],2)] ^ m[shared_message_index_64];		
        break;
      case 2:
      m[shared_message_index_64] = T_m64[3*256+EXT_BYTE(x[ 2],3)] ^ m[shared_message_index_64];		
        break;
      case 0:
        m[shared_message_index_64] = T_m64[0*256+EXT_BYTE(x[ 2],0)] ^ m[shared_message_index_64]; 
        break;
   }
									
    __syncthreads();

}

// #define ROUNDQ512(m_in, m, r, T_shared, sharedMessageIndex ) do {				\	
//     u64* T_m64 = (u64*)T_shared;						\
//     u64 zero;								\
//     u64 ff;								\
//     u32* x = (u32*)m_in;						\
//     zero = 0;						\
//     ff   = -1;				\
//     int localMessageBlockIndex = sharedMessageIndex % COLS512; \
// 									\
//     m_in[localMessageBlockIndex] = (m_in[localMessageBlockIndex]^ff);					\
//     // m_in[1] = (m_in[1]^ff);					\
//     // m_in[2] = (m_in[2]^ff);					\
//     // m_in[3] = (m_in[3]^ff);					\
//     // m_in[4] = (m_in[4]^ff);					\
//     // m_in[5] = (m_in[5]^ff);					\
//     // m_in[6] = (m_in[6]^ff);					\
//     // m_in[7] = (m_in[7]^ff);					
//     (localMessageBlockIndex == 7) 

//     m[7] = T_m64[0*256+EXT_BYTE(x[ 0],0)] ^ zero;		\
//     m[5] = T_m64[1*256+EXT_BYTE(x[ 0],1)] ^ zero;		\
//     m[3] = T_m64[2*256+EXT_BYTE(x[ 0],2)] ^ zero;		\
//     m[1] = T_m64[3*256+EXT_BYTE(x[ 0],3)] ^ zero;		\
//     m[0] = T_m64[4*256+EXT_BYTE(x[ 1],0)] ^ zero;		\
//     m[6] = T_m64[5*256+EXT_BYTE(x[ 1],1)] ^ zero;		\
//     m[4] = T_m64[6*256+EXT_BYTE(x[ 1],2)] ^ zero;		\
//     m[2] = T_m64[7*256+((r)^EXT_BYTE(x[ 1],3))] ^ zero;	\
// 									\
//     m[6] = T_m64[0*256+EXT_BYTE(x[14],0)] ^ m[6];		\
//     m[4] = T_m64[1*256+EXT_BYTE(x[14],1)] ^ m[4];		\
//     m[2] = T_m64[2*256+EXT_BYTE(x[14],2)] ^ m[2];		\
//     m[0] = T_m64[3*256+EXT_BYTE(x[14],3)] ^ m[0];		\
//     m[7] = T_m64[4*256+EXT_BYTE(x[15],0)] ^ m[7];		\
//     m[5] = T_m64[5*256+EXT_BYTE(x[15],1)] ^ m[5];		\
//     m[3] = T_m64[6*256+EXT_BYTE(x[15],2)] ^ m[3];		\
//     m[1] = T_m64[7*256+((r)^0x70^EXT_BYTE(x[15],3))] ^ m[1]; \
// 									\
//     m[5] = T_m64[0*256+EXT_BYTE(x[12],0)] ^ m[5];		\
//     m[3] = T_m64[1*256+EXT_BYTE(x[12],1)] ^ m[3];		\
//     m[1] = T_m64[2*256+EXT_BYTE(x[12],2)] ^ m[1];		\
//     m[7] = T_m64[3*256+EXT_BYTE(x[12],3)] ^ m[7];		\
//     m[6] = T_m64[4*256+EXT_BYTE(x[13],0)] ^ m[6];		\
//     m[4] = T_m64[5*256+EXT_BYTE(x[13],1)] ^ m[4];		\
//     m[2] = T_m64[6*256+EXT_BYTE(x[13],2)] ^ m[2];		\
//     m[0] = T_m64[7*256+((r)^0x60^EXT_BYTE(x[13],3))] ^ m[0]; \
// 									\
//     m[4] = T_m64[0*256+EXT_BYTE(x[10],0)] ^ m[4];		\
//     m[2] = T_m64[1*256+EXT_BYTE(x[10],1)] ^ m[2];		\
//     m[0] = T_m64[2*256+EXT_BYTE(x[10],2)] ^ m[0];		\
//     m[6] = T_m64[3*256+EXT_BYTE(x[10],3)] ^ m[6];		\
//     m[5] = T_m64[4*256+EXT_BYTE(x[11],0)] ^ m[5];		\
//     m[3] = T_m64[5*256+EXT_BYTE(x[11],1)] ^ m[3];		\
//     m[1] = T_m64[6*256+EXT_BYTE(x[11],2)] ^ m[1];		\
//     m[7] = T_m64[7*256+((r)^0x50^EXT_BYTE(x[11],3))] ^ m[7]; \
// 									\
//     m[3] = T_m64[0*256+EXT_BYTE(x[ 8],0)] ^ m[3];		\
//     m[1] = T_m64[1*256+EXT_BYTE(x[ 8],1)] ^ m[1];		\
//     m[7] = T_m64[2*256+EXT_BYTE(x[ 8],2)] ^ m[7];		\
//     m[5] = T_m64[3*256+EXT_BYTE(x[ 8],3)] ^ m[5];		\
//     m[4] = T_m64[4*256+EXT_BYTE(x[ 9],0)] ^ m[4];		\
//     m[2] = T_m64[5*256+EXT_BYTE(x[ 9],1)] ^ m[2];		\
//     m[0] = T_m64[6*256+EXT_BYTE(x[ 9],2)] ^ m[0];		\
//     m[6] = T_m64[7*256+((r)^0x40^EXT_BYTE(x[ 9],3))] ^ m[6]; \
// 									\
//     m[2] = T_m64[0*256+EXT_BYTE(x[ 6],0)] ^ m[2];		\
//     m[0] = T_m64[1*256+EXT_BYTE(x[ 6],1)] ^ m[0];		\
//     m[6] = T_m64[2*256+EXT_BYTE(x[ 6],2)] ^ m[6];		\
//     m[4] = T_m64[3*256+EXT_BYTE(x[ 6],3)] ^ m[4];		\
//     m[3] = T_m64[4*256+EXT_BYTE(x[ 7],0)] ^ m[3];		\
//     m[1] = T_m64[5*256+EXT_BYTE(x[ 7],1)] ^ m[1];		\
//     m[7] = T_m64[6*256+EXT_BYTE(x[ 7],2)] ^ m[7];		\
//     m[5] = T_m64[7*256+((r)^0x30^EXT_BYTE(x[ 7],3))] ^ m[5]; \
// 									\
//     m[1] = T_m64[0*256+EXT_BYTE(x[ 4],0)] ^ m[1];		\
//     m[7] = T_m64[1*256+EXT_BYTE(x[ 4],1)] ^ m[7];		\
//     m[5] = T_m64[2*256+EXT_BYTE(x[ 4],2)] ^ m[5];		\
//     m[3] = T_m64[3*256+EXT_BYTE(x[ 4],3)] ^ m[3];		\
//     m[2] = T_m64[4*256+EXT_BYTE(x[ 5],0)] ^ m[2];		\
//     m[0] = T_m64[5*256+EXT_BYTE(x[ 5],1)] ^ m[0];		\
//     m[6] = T_m64[6*256+EXT_BYTE(x[ 5],2)] ^ m[6];		\
//     m[4] = T_m64[7*256+((r)^0x20^EXT_BYTE(x[ 5],3))] ^ m[4]; \
// 									\
//     m[0] = T_m64[0*256+EXT_BYTE(x[ 2],0)] ^ m[0];		\
//     m[6] = T_m64[1*256+EXT_BYTE(x[ 2],1)] ^ m[6];		\
//     m[4] = T_m64[2*256+EXT_BYTE(x[ 2],2)] ^ m[4];		\
//     m[2] = T_m64[3*256+EXT_BYTE(x[ 2],3)] ^ m[2];		\
//     m[1] = T_m64[4*256+EXT_BYTE(x[ 3],0)] ^ m[1];		\
//     m[7] = T_m64[5*256+EXT_BYTE(x[ 3],1)] ^ m[7];		\
//     m[5] = T_m64[6*256+EXT_BYTE(x[ 3],2)] ^ m[5];		\
//     m[3] = T_m64[7*256+((r)^0x10^EXT_BYTE(x[ 3],3))] ^ m[3]; \
//   } while (0)



// typedef struct {
//     // hashState *ctx;        // Pointer to the hash state
//     const u8 *msg_block;   // Pointer to the current message block
//     u64 *resultBlock ;
//     pthread_t thread_id;        // Thread ID
//     // ... any other arguments needed for processing ...
// } ThreadArgs;


// ThreadArgs *setup_thread_args(ThreadArgs *args,  const u8 *msg_block) {
//     if (msg_block == NULL) {
//         // Handle memory allocation failure
//         printf('msg block nill');
//     }
//     // args->ctx = ctx;
//     args->msg_block = msg_block;
//     args-> resultBlock = malloc(COLS512 * sizeof(u64));
//     return args;
// }


// void ProcessBlock(ThreadArgs* args) {
//     u64 *m64_m = args->resultBlock, tmp[COLS512];
//     u64 *msg_64 = (u64*)args->msg_block;

//     for (int i = 0; i < COLS512; i++) {
//       m64_m[i] = msg_64[i];
//       // m64_hm[i] = m64_h[i] ^ m64_m[i];
//     }

//     // Perform the ROUNDQ512 operations
//     ROUNDQ512(m64_m, tmp, 0);
//     ROUNDQ512(tmp, m64_m, 1);
//     ROUNDQ512(m64_m, tmp, 2);
//     ROUNDQ512(tmp, m64_m, 3);
//     ROUNDQ512(m64_m, tmp, 4);
//     ROUNDQ512(tmp, m64_m, 5);
//     ROUNDQ512(m64_m, tmp, 6);
//     ROUNDQ512(tmp, m64_m, 7);
//     ROUNDQ512(m64_m, tmp, 8);
//     ROUNDQ512(tmp, m64_m, 9);

//     // ... and so on for the rest of the ROUNDQ512 calls ...

//     // // Final combination steps (similar to the end of the while loop in Transform512)
//     // for (int i = 0; i < COLS512; i++) {
//     //     // ... Same combination steps as in Transform512 ...
//     // }
// }

/* apply the output transformation after identifying variant */
// __device__ void OutputTransformation(u32 *output, u64 *T_shared) {
//     OutputTransformation512(output, T_shared);
// }


/* digest part of a message in short variants */
__global__ void Transform512(u32 *outputTransformation, int outputTransformSize, const u8 *msg, int msglen, const u8*  msg_transformation_buffer)  {

   printf("Transform512\n");
   return;
   
   int i;
  __shared__ u64 m64_m[COLS512];
  __shared__ u64 *m64_h;
  __shared__ u64 m64_hm[COLS512];
  __shared__ u64 tmp[COLS512];
  __shared__ u64 tmp_output[COLS512];
  __shared__ u64 output_transformation_shared[COLS512];

  // u8 *tmp_msg_transform;
  // if (blockIdx.x && threadId.x == 0) {
  //     tmp_msg_transform = (u8*)malloc(msglen);
  // }

  __syncthreads(); // Synchronize to ensure the allocation is done

  int threadsPerBlock = blockDim.x;
  int totalLoads = 2048 / threadsPerBlock; // Each thread loads this many bytes

  int sharedMemoryLoadIndex = threadIdx.x * totalLoads;
  int end = sharedMemoryLoadIndex + totalLoads;

  #define TABLE_SIZE 2048
  __shared__ u64 T_shared[TABLE_SIZE];
  // Load data into shared memory
  for (int runningLoadIndex = sharedMemoryLoadIndex; runningLoadIndex < end; ++runningLoadIndex) {
      // sharedData[runningLoadIndex] = globalData[runningLoadIndex];
      if (runningLoadIndex < TABLE_SIZE) { // Ensure we don't go out of bounds of shared memory
        printf("runningLoadIndex:%d\n", runningLoadIndex);
        T_shared[runningLoadIndex] = T[runningLoadIndex];
      }
  }

  // Synchronize to ensure all data is loaded
  __syncthreads();

  int grid_stride_local_index = blockIdx.x * blockDim.x + threadIdx.x; 
  int stride = blockDim.x * gridDim.x;
  u64 *msg_64 = (u64*)msg;

  #define SHARED_Q_RESULTS_SIZE 512 
  
  __shared__ u64 q_shared_results_64[SHARED_Q_RESULTS_SIZE];
  // __shared__ int index_q_shared_results_64;
  __shared__ u64 tmp_shared_64[SHARED_Q_RESULTS_SIZE];
  __shared__ u64 p_shared_results_64[SHARED_Q_RESULTS_SIZE];

  if (blockDim.x > SHARED_Q_RESULTS_SIZE ){
      printf("we have an error, block dimensions cannot be bigger than shared q results size  ");
      return;
  }

  u64* transformation_buffer_q_64 = (u64*)msg_transformation_buffer;
  int msglen_64 = msglen/8; // length of char* data type to  u64* type

  for (int global_index_64 = grid_stride_local_index; global_index_64 < (msglen_64 + stride); global_index_64 += stride) {
      if (global_index_64 < msglen_64){
        int q_local_index_shared_64 = global_index_64 % SHARED_Q_RESULTS_SIZE;
        q_shared_results_64[q_local_index_shared_64] = msg_64[global_index_64];
        // p_shared_results_64[q_local_index_shared_64] = msg_64[global_index_64] ^  ;
        
        ROUNDQ512_function(q_shared_results_64, tmp_shared_64, 0,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(tmp_shared_64, q_shared_results_64, 1,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(q_shared_results_64, tmp_shared_64, 2,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(tmp_shared_64, q_shared_results_64, 3,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(q_shared_results_64, tmp_shared_64, 4,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(tmp_shared_64, q_shared_results_64, 5,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(q_shared_results_64, tmp_shared_64, 6,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(tmp_shared_64, q_shared_results_64, 7,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(q_shared_results_64, tmp_shared_64, 8,(u64*) T_shared, q_local_index_shared_64);
        ROUNDQ512_function(tmp_shared_64, q_shared_results_64, 9,(u64*) T_shared, q_local_index_shared_64);

        transformation_buffer_q_64[global_index_64] = q_shared_results_64[q_local_index_shared_64];
      }
  }

  // thread coarsening to 8 total threads
  if (blockIdx.x == 0 && threadIdx.x < 8){
    int threadId = threadIdx.x;
    m64_h = (u64*)outputTransformation;
    output_transformation_shared[threadId] = outputTransformation[threadId];

    while (msglen >= SIZE512) {
        msg_64 = (u64*)msg;

        // for (i = 0; i < COLS512; i++) {
          // printf("i:%d\n", i);
          // m64_m[i] = msg_64[i];
        // q_shared_results_64[threadId] = msg_64[threadId];
        m64_hm[threadId] = output_transformation_shared[threadId] ^ msg_64[threadId];
        // }

        ROUNDP512_function(m64_hm, tmp, 0, T_shared, threadId);
        ROUNDP512_function(tmp, m64_hm, 1, T_shared, threadId);
        ROUNDP512_function(m64_hm, tmp, 2, T_shared, threadId);
        ROUNDP512_function(tmp, m64_hm, 3, T_shared, threadId);
        ROUNDP512_function(m64_hm, tmp, 4, T_shared, threadId);
        ROUNDP512_function(tmp, m64_hm, 5, T_shared, threadId);
        ROUNDP512_function(m64_hm, tmp, 6, T_shared, threadId);
        ROUNDP512_function(tmp, m64_hm, 7, T_shared, threadId);
        ROUNDP512_function(m64_hm, tmp, 8, T_shared, threadId);
        ROUNDP512_function(tmp, m64_hm, 9, T_shared, threadId);

        // ROUNDQ512(m64_m, tmp, 0, T_shared);
        // ROUNDQ512(tmp, m64_m, 1, T_shared);
        // ROUNDQ512(m64_m, tmp, 2, T_shared);
        // ROUNDQ512(tmp, m64_m, 3, T_shared);
        // ROUNDQ512(m64_m, tmp, 4, T_shared);
        // ROUNDQ512(tmp, m64_m, 5, T_shared);
        // ROUNDQ512(m64_m, tmp, 6, T_shared);
        // ROUNDQ512(tmp, m64_m, 7, T_shared);
        // ROUNDQ512(m64_m, tmp, 8, T_shared);
        // ROUNDQ512(tmp, m64_m, 9, T_shared);
      
        // m64_m = threads_args[block_index].resultBlock;
        
        // for (i = 0; i < COLS512; i++) {
        output_transformation_shared[threadId] = output_transformation_shared[threadId] ^  msg_transformation_buffer [msglen/8 + threadId];
        output_transformation_shared[threadId] = output_transformation_shared[threadId] ^ m64_hm[threadId];
        // }
        // for (i = 0; i < COLS512; i++) {
        //   m64_h[i] = m64_h[i] ^ threads_args[block_index].resultBlock[i];
        //   m64_h[i] = m64_h[i] ^ m64_hm[i];
        // }

        msg += SIZE512;
        msglen -= SIZE512;    
        // block_index++;

      }

      __syncthreads();

      // OutputTransformation(output_transformation_shared, (u64 *)T_shared);

      // int i;
      // u64 *m64_h, tmp1[COLS512], tmp2[COLS512];
      // m64_h = (u64*)outputTransformation;

      // for (i = 0; i < COLS512; i++) {
      tmp[threadId] = output_transformation_shared[threadId];
      // }

      ROUNDP512_function(tmp, tmp_output, 0, T_shared, threadId);
      ROUNDP512_function(tmp_output, tmp, 1, T_shared, threadId);
      ROUNDP512_function(tmp, tmp_output, 2, T_shared, threadId);
      ROUNDP512_function(tmp_output, tmp, 3, T_shared, threadId);
      ROUNDP512_function(tmp, tmp_output, 4, T_shared, threadId);
      ROUNDP512_function(tmp_output, tmp, 5, T_shared, threadId);
      ROUNDP512_function(tmp, tmp_output, 6, T_shared, threadId);
      ROUNDP512_function(tmp_output, tmp, 7, T_shared, threadId);
      ROUNDP512_function(tmp, tmp_output, 8, T_shared, threadId);
      ROUNDP512_function(tmp_output, tmp, 9, T_shared, threadId);

      // for (i = 0; i < COLS512; i++) {
      m64_h[threadId] = output_transformation_shared[threadId] ^ tmp[threadId];
      // }

      return;
  }

  // __shared__ u64 T_shared[8 * 256];

  // if (tid < 4 * 256) {
  //     T_shared[tid*2] = T[tid*2];
  //     T_shared[tid*2 + 1] = T[tid*2 + 1];
  // }
  // __syncthreads();

  // u64 *T_shared = (u64 *)T;

  // Determine the number of blocks
  // int num_blocks = msglen / SIZE512;
  //  pthread_t threads[num_blocks];
  // ThreadArgs threads_args[num_blocks];

  // int block_index = 0;

  // for (block_index = 0; block_index < num_blocks; block_index++) {
  //   // Arguments for ProcessBlock function
  //   setup_thread_args(&threads_args[block_index], msg + (block_index * SIZE512));
  //   pthread_create(&threads_args[block_index].thread_id, NULL, ProcessBlock, &threads_args[block_index]);
  // }

  // // Wait for all threads to complete
  // for (block_index = 0;block_index < num_blocks; block_index++) {
  //   pthread_join(threads_args[block_index].thread_id, NULL);
  // }

  // block_index = 0;

}

__global__ void TestTransform(){
  printf("test transform \n");
}

/* digest part of a message after identifying variant */
int Transform(u32 *outputTransformation, int outputTransformSize, const u8 *msg, int msglen) {

    u8 * deviceMessage, *deviceOutputTransform, *deviceTransformationBuffer;
    // uint size = n*sizeof(uint);
    struct timeval start, end;
    u64 usec;

    hipDeviceSynchronize();
    checkCudaErrors(hipMalloc((void**)&deviceMessage, msglen));
    checkCudaErrors(hipMalloc((void**)&deviceTransformationBuffer, msglen)); // to store the q results on device.
    checkCudaErrors(hipMalloc((void**)&deviceOutputTransform, outputTransformSize));
    gettimeofday(&start, NULL);
    checkCudaErrors(hipMemcpy(deviceMessage, msg, msglen, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(deviceOutputTransform, outputTransformation, outputTransformSize, hipMemcpyHostToDevice));
    gettimeofday(&end, NULL);
    usec = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
    printf("HtoD %lld usec\n", usec);

    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    dim3 dimBlock(512, 1, 1);
    dim3 dimGrid(8, 1, 1);

    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(MyKernel), hipFuncCachePreferShared);
    //hipFuncSetCacheConfig(reinterpret_cast<const void*>(AES_encrypt), hipFuncCachePreferL1);

    // warmup
    // AES_encrypt<<<dimGrid, dimBlock>>>(cpt, cct, ce_sched, Nr, n);

    // debug<<<1, 1>>>();
    getLastCudaError();

    for (int i = 0; i < 1; i++) {
        hipDeviceSynchronize();
        gettimeofday(&start, NULL);
        // Transform512<<<dimGrid, dimBlock>>>((u32 *)deviceOutputTransform, outputTransformSize, (u8*)deviceMessage, msglen, deviceTransformationBuffer );
        TestTransform<<<dimGrid, dimBlock>>>();
        // AES_encrypt<<<dimGrid, dimBlock>>>(cpt, cct, ce_sched, Nr, n);
        hipDeviceSynchronize();
        //exit(0);
        getLastCudaError();
        gettimeofday(&end, NULL);
        usec = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
        u64 size_in_MB = msglen / 1024 / 1024;
        const char *gors = "smem" ;
        printf(" %s %d MB %lld usec %lf Gbps\n", 
                 gors, size_in_MB, usec, 
                ((double)size_in_MB*8/1024) / ((double)usec/1000000));
    }
    gettimeofday(&start, NULL);
    checkCudaErrors(hipMemcpy(outputTransformation, deviceOutputTransform, outputTransformSize, hipMemcpyDeviceToHost));
    gettimeofday(&end, NULL);
    usec = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
    printf("DtoH %lld usec\n", usec);
	
    checkCudaErrors(hipFree(deviceMessage));
    checkCudaErrors(hipFree(deviceOutputTransform));
    hipDeviceSynchronize();
    return 1;

}

// /* apply the output transformation of short variants */
// __device__ void OutputTransformation512(u32 *outputTransformation, u64* T_shared) {
//   int i;
//   u64 *m64_h, tmp1[COLS512], tmp2[COLS512];
//   m64_h = (u64*)outputTransformation;

//   for (i = 0; i < COLS512; i++) {
//     tmp1[i] = m64_h[i];
//   }

//   ROUNDP512(tmp1, tmp2, 0, T_shared);
//   ROUNDP512(tmp2, tmp1, 1, T_shared);
//   ROUNDP512(tmp1, tmp2, 2, T_shared);
//   ROUNDP512(tmp2, tmp1, 3, T_shared);
//   ROUNDP512(tmp1, tmp2, 4, T_shared);
//   ROUNDP512(tmp2, tmp1, 5, T_shared);
//   ROUNDP512(tmp1, tmp2, 6, T_shared);
//   ROUNDP512(tmp2, tmp1, 7, T_shared);
//   ROUNDP512(tmp1, tmp2, 8, T_shared);
//   ROUNDP512(tmp2, tmp1, 9, T_shared);

//   for (i = 0; i < COLS512; i++) {
//     m64_h[i] = m64_h[i] ^ tmp1[i];
//   }
// }



/* initialise context */
HashReturn Init(hashState* ctx,
		int hashbitlen) {
  /* output size (in bits) must be a positive integer less than or
     equal to 512, and divisible by 8 */
  if (hashbitlen <= 0 || (hashbitlen%8) || hashbitlen > 512)
    return BAD_HASHLEN;

  /* set number of state columns and state size depending on
     variant */
  if (hashbitlen <= 256) {
    ctx->columns = COLS512;
    ctx->statesize = SIZE512;
  }
  else {
    // ctx->columns = COLS1024;
    // ctx->statesize = SIZE1024;
  }



  /* set other variables */
  ctx->hashbitlen = hashbitlen;
  ctx->block_counter = 0;

  return SUCCESS;
}

/* update state with databitlen bits of input */
HashReturn Update(hashState* ctx,
		  const BitSequence* input,
		  DataLength databitlen, u32* transformedOutput) {
  int index = 0;
  const int msglen = (int)(databitlen/8);
  int newMsgLen = msglen;
  int rem = (int)(databitlen%8);
  uchar* byteInput = (uchar*)input;

  ctx->block_counter = msglen / ctx->statesize;
  byteInput[newMsgLen] = 0x80;
  newMsgLen++;

  const int remainder = (newMsgLen)%ctx->statesize;
  int remainderIndex = remainder;
  /* store remaining data in buffer */
  if (remainderIndex > ctx->statesize - LENGTHFIELDLEN) {
    // extra buffer
    while (remainderIndex < ctx->statesize) {
      byteInput[newMsgLen] = 0;
      remainderIndex++;
      newMsgLen++;
    }
    // newMsgLen = newMsgLen + (ctx->statesize - remainder);
    remainderIndex = 0;
    ctx->block_counter++;
  }

  while (remainderIndex < ctx->statesize-LENGTHFIELDLEN) {
    byteInput[newMsgLen] = 0;
    remainderIndex++;
    newMsgLen++;
  }
  ctx->block_counter++;

  // byteInput[newMsgLen + (remainderIndex -1 )] = (u8)ctx->block_counter;
  newMsgLen += LENGTHFIELDLEN;

  int lengthPad = LENGTHFIELDLEN;
  int lengthPadIndex = 1;
  while (lengthPadIndex <= LENGTHFIELDLEN) {
    byteInput[newMsgLen - lengthPadIndex] = (u8)ctx->block_counter;
    lengthPadIndex++;
    ctx->block_counter >>= 8;
  }

  Transform(transformedOutput, ctx->statesize, input, newMsgLen);
  return SUCCESS;
}


/* finalise: process remaining data (including padding), perform
   output transformation, and write hash result to 'output' */
HashReturn Final(hashState* ctx, u32* input,
		 BitSequence* output) {
  int i, j = 0, hashbytelen = ctx->hashbitlen/8;
  u8 *s = (u8*)input;

  /* store hash result in output */
  for (i = ctx->statesize-hashbytelen; i < ctx->statesize; i++,j++) {
    output[j] = s[i];
  }

  /* zeroise relevant variables and deallocate memory */
  for (i = 0; i < ctx->columns; i++) {
    input[i] = 0;
  }
  // free(ctx->chaining);
  // free(ctx->buffer);
  return SUCCESS;
}

/* hash bit sequence */
HashReturn Hash(int hashbitlen,
		const BitSequence* data, 
		DataLength databitlen,
		BitSequence* hashval) {
  HashReturn ret;
  hashState context;

  /* initialise */
  if ((ret = Init(&context, hashbitlen)) != SUCCESS)
    return ret;
    
  u32* transformedOutput =(u32*) calloc(context.statesize,1);
  /* allocate memory for state and data buffer */
  transformedOutput[2*context.columns-1] = U32BIG((u32)context.hashbitlen);

  /* process message */
  if ((ret = Update(&context, data, databitlen, transformedOutput)) != SUCCESS)
    return ret;

  /* finalise */
  ret = Final(&context, transformedOutput, hashval);


  free(transformedOutput);

  return ret;
}

void PrintHash(const BitSequence* hash,
	       int hashbitlen) {
  int i;
  for (i = 0; i < hashbitlen/8; i++) {
    printf("%02x", hash[i]);
  }
  printf("\n");
}

// /* eBash API */
#define crypto_hash_BYTES 32u
#ifdef crypto_hash_BYTES
int crypto_hash(unsigned char *out, const unsigned char *in, u64 inlen)
{
  if (Hash(256, in, inlen * 8,out) == SUCCESS) return 0;
  return -1;
}
#endif

void printHexArray(unsigned char *array, uint size) {
    int i;
    for(i=0 ; i< size; i++)
    	printf("%02x", array[i]);
    printf("\n");
}


int main(int argc, char **argv) {
    uint *ct, *pt;
    ct = (uint*)malloc(8 * sizeof(uint)); // Allocating memory for 8 uints

    int dataSize; // Total data size
    size_t maxSharedMemory;
    FILE *file = fopen("groestl256.blb", "r");

    if (file == NULL) {
      printf("Error opening the file.\n");
      return -1;
    }

    fseek(file, 0, SEEK_END);
    dataSize = ftell(file);
    fseek(file, 0, SEEK_SET);

    // Host array
    unsigned char *hostData = (unsigned char*)malloc(dataSize + (SIZE512 * 2));
    if (hostData == NULL) {
      printf("Error allocating memory.\n");
      fclose(file);
      return -1;
    }

    fread(hostData, sizeof(unsigned char), dataSize, file);
    fclose(file);

    const char* message = "my message gdfjhghjkfdhgjklfdshgjklfdhgjkfdshkfjsdhgjfdlshgjkfdsghfjdklhgjfkdlghfjdkslhgfdjksgsdfhj    dsdscxcd3232322cc";
    size_t size = strlen(message);

    unsigned char* data = (unsigned char*)malloc(size + (SIZE512 * 2));
    memcpy(data, message, size);
    crypto_hash((uchar*)ct, data, size);

    // printf("Data: %s\n", hostData);
    // printf("Size: %zu\n", dataSize);
    // crypto_hash(ct, hostData, dataSize);

    printHexArray((uchar*)ct, 32);
    printf("done done\n");
    return 1;
}


#else
#error "MMX instructions must be enabled"
#endif /* __MMX__ */
