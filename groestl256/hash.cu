#include "hip/hip_runtime.h"
/*
 * CUDA-fied by Fawad Haider January 2024
 *
 */

/* hash.c   January 2011
 *
 * Groestl implementation using MMX intrinsics.
 * Has been tested on Intel processors using the gcc compiler only
 * (include the '-std=c99' and '-mmmx' flags when compiling in gcc on a
 * 32-bit machine).
 *
 * Author: Soeren S. Thomsen
 *
 * This code is placed in the public domain
 */
#define __MMX__ 81
#define CUDAFIED 80
#if defined(__MMX__)
#include <stdio.h>
#include <stdlib.h>
#include "hash.h"
#include "tables.h"
#include <string.h>
#include <pthread.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

void check(hipError_t err, const char *const func, const char *const file,
           const int line)
{
  if (err != hipSuccess)
  {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line
              << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    // We don't exit when we encounter CUDA errors in this example.
    // std::exit(EXIT_FAILURE);
  }
}

#define getLastCudaError() checkLast(__FILE__, __LINE__)

void checkLast(const char *const file, const int line)
{
  hipError_t const err{hipGetLastError()};
  if (err != hipSuccess)
  {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line
              << std::endl;
    std::cerr << hipGetErrorString(err) << std::endl;
    // We don't exit when we encounter CUDA errors in this example.
    // std::exit(EXIT_FAILURE);
  }
}

__device__ void ROUNDP512(u64 *m_in, u64 *m, int r, u64 *T_shared, int shared_message_index_64)
{

  u64 *T_m64 = (u64 *)T_shared;
  u64 zero;

  int local_message_block_index_32 = shared_message_index_64 % COLS512;
  int start_message_index = shared_message_index_64 - local_message_block_index_32; // need to find the start offset for the current message block

  u32 *x = (u32 *)&m_in[start_message_index];
  zero = 0;

  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[0], 3)] ^ zero;
    break;
  case 3:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[1], 1)] ^ zero;
    break;
  case 1:
    m[shared_message_index_64] = T_m64[7 * 256 + EXT_BYTE(x[1], 3)] ^ zero;
    break;
  case 7:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[0], 1)] ^ zero;
    break;
  case 6:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[0], 2)] ^ zero;
    break;
  case 4:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[1], 0)] ^ zero;
    break;
  case 2:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[1], 2)] ^ zero;
    break;
  case 0:
    m[shared_message_index_64] = T_m64[0 * 256 + ((r) ^ EXT_BYTE(x[0], 0))] ^ zero;
    break;
  }
  //  __syncthreads();
  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[14], 2)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[15], 0)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[15], 2)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[0 * 256 + ((r) ^ 0x70 ^ EXT_BYTE(x[14], 0))] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[14], 1)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[14], 3)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[15], 1)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[7 * 256 + EXT_BYTE(x[15], 3)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();
  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[12], 1)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[12], 3)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[13], 1)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[7 * 256 + EXT_BYTE(x[13], 3)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[0 * 256 + ((r) ^ 0x60 ^ EXT_BYTE(x[12], 0))] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[12], 2)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[13], 0)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[13], 2)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();
  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[0 * 256 + ((r) ^ 0x50 ^ EXT_BYTE(x[10], 0))] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[10], 2)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[11], 0)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[11], 2)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[7 * 256 + EXT_BYTE(x[11], 3)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[10], 1)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[10], 3)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[11], 1)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();
  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[7 * 256 + EXT_BYTE(x[9], 3)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[8], 1)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[8], 3)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[9], 1)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[9], 2)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[0 * 256 + ((r) ^ 0x40 ^ EXT_BYTE(x[8], 0))] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[8], 2)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[9], 0)] ^ m[shared_message_index_64];
    break;
  }

 // __syncthreads();
  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[7], 2)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[0 * 256 + ((r) ^ 0x30 ^ EXT_BYTE(x[6], 0))] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[6], 2)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[7], 0)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[7], 1)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[7 * 256 + EXT_BYTE(x[7], 3)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[6], 1)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[6], 3)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();
  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[5], 1)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[7 * 256 + EXT_BYTE(x[5], 3)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[4], 1)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[4], 3)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[5], 0)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[5], 2)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[0 * 256 + ((r) ^ 0x20 ^ EXT_BYTE(x[4], 0))] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[4], 2)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();
  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[3], 0)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[3], 2)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[0 * 256 + ((r) ^ 0x10 ^ EXT_BYTE(x[2], 0))] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[2], 2)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[2], 3)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[3], 1)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[7 * 256 + EXT_BYTE(x[3], 3)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[2], 1)] ^ m[shared_message_index_64];
    break;
  }

 // __syncthreads();
}

__device__ void ROUNDQ512(u64 *m_in, u64 *m, int r, u64 *T_shared, int shared_message_index_64)
{

  u64 *T_m64 = (u64 *)T_shared;
  u64 zero;
  u64 ff;
  int local_message_block_index_32 = shared_message_index_64 % COLS512;
  int start_message_index = shared_message_index_64 - local_message_block_index_32; // need to find the start offset for the current message block
  u32 *x = (u32 *)&m_in[start_message_index];
  zero = 0;
  ff = -1;

  m_in[shared_message_index_64] = (m_in[shared_message_index_64] ^ ff); // even is for shared_message_index_64 64 bits array, we can use it in the 2 bit array in this case

  switch (local_message_block_index_32)
  {
  case 7:
    m[shared_message_index_64] = T_m64[0 * 256 + EXT_BYTE(x[0], 0)] ^ zero;
    break;

  case 5:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[0], 1)] ^ zero;
    break;

  case 3:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[0], 2)] ^ zero;
    break;

  case 1:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[0], 3)] ^ zero;
    break;

  case 0:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[1], 0)] ^ zero;
    break;

  case 6:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[1], 1)] ^ zero;
    break;

  case 4:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[1], 2)] ^ zero;
    break;

  case 2:
    m[shared_message_index_64] = T_m64[7 * 256 + ((r) ^ EXT_BYTE(x[1], 3))] ^ zero;
    break;
  }

  // __syncthreads();

  switch (local_message_block_index_32)
  {
  case 7:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[15], 0)] ^ m[shared_message_index_64];
    break;

  case 5:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[15], 1)] ^ m[shared_message_index_64];
    break;

  case 3:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[15], 2)] ^ m[shared_message_index_64];
    break;

  case 1:
    m[shared_message_index_64] = T_m64[7 * 256 + ((r) ^ 0x70 ^ EXT_BYTE(x[15], 3))] ^ m[shared_message_index_64];
    break;

  case 0:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[14], 3)] ^ m[shared_message_index_64];
    break;

  case 6:
    m[shared_message_index_64] = T_m64[0 * 256 + EXT_BYTE(x[14], 0)] ^ m[shared_message_index_64];
    break;

  case 4:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[14], 1)] ^ m[shared_message_index_64];
    break;

  case 2:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[14], 2)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();

  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[0 * 256 + EXT_BYTE(x[12], 0)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[12], 1)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[12], 2)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[12], 3)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[13], 0)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[13], 1)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[13], 2)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[7 * 256 + ((r) ^ 0x60 ^ EXT_BYTE(x[13], 3))] ^ m[shared_message_index_64];
    break;
  }
  // __syncthreads();

  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[11], 0)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[11], 1)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[11], 2)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[7 * 256 + ((r) ^ 0x50 ^ EXT_BYTE(x[11], 3))] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[10], 3)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[0 * 256 + EXT_BYTE(x[10], 0)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[10], 1)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[10], 2)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();

  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[8], 3)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[0 * 256 + EXT_BYTE(x[8], 0)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[8], 1)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[8], 2)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[7 * 256 + ((r) ^ 0x40 ^ EXT_BYTE(x[9], 3))] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[9], 0)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[9], 1)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[9], 2)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();

  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[7 * 256 + ((r) ^ 0x30 ^ EXT_BYTE(x[7], 3))] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[7], 0)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[7], 1)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[7], 2)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[6], 2)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[6], 3)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[0 * 256 + EXT_BYTE(x[6], 0)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[6], 1)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();

  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[4], 2)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[4], 3)] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[0 * 256 + EXT_BYTE(x[4], 0)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[4], 1)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[5], 2)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[7 * 256 + ((r) ^ 0x20 ^ EXT_BYTE(x[5], 3))] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[5], 0)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[5], 1)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();

  switch (local_message_block_index_32)
  {
  case 5:
    m[shared_message_index_64] = T_m64[6 * 256 + EXT_BYTE(x[3], 2)] ^ m[shared_message_index_64];
    break;
  case 3:
    m[shared_message_index_64] = T_m64[7 * 256 + ((r) ^ 0x10 ^ EXT_BYTE(x[3], 3))] ^ m[shared_message_index_64];
    break;
  case 1:
    m[shared_message_index_64] = T_m64[4 * 256 + EXT_BYTE(x[3], 0)] ^ m[shared_message_index_64];
    break;
  case 7:
    m[shared_message_index_64] = T_m64[5 * 256 + EXT_BYTE(x[3], 1)] ^ m[shared_message_index_64];
    break;
  case 6:
    m[shared_message_index_64] = T_m64[1 * 256 + EXT_BYTE(x[2], 1)] ^ m[shared_message_index_64];
    break;
  case 4:
    m[shared_message_index_64] = T_m64[2 * 256 + EXT_BYTE(x[2], 2)] ^ m[shared_message_index_64];
    break;
  case 2:
    m[shared_message_index_64] = T_m64[3 * 256 + EXT_BYTE(x[2], 3)] ^ m[shared_message_index_64];
    break;
  case 0:
    m[shared_message_index_64] = T_m64[0 * 256 + EXT_BYTE(x[2], 0)] ^ m[shared_message_index_64];
    break;
  }

  // __syncthreads();
}

#define TABLE_SIZE 2048
#define SHARED_Q_RESULTS_SIZE 512

/* digest part of a message in short variants */
__global__ void Transform512(u32 *outputTransformation, int outputTransformSize, const u8 *msg, int msglen, const u8 *msg_transformation_buffer)
{

  int i;

  //TODO some of these shared mem vars can be re-used and redced 
  __shared__ u64 m64_m[COLS512];
  __shared__ u64 *m64_h;
  __shared__ u64 m64_hm[COLS512];
  __shared__ u64 tmp[COLS512];
  __shared__ u64 tmp_output[COLS512];
  __shared__ u64 output_transformation_shared[COLS512];
  __shared__ u64 T_shared[TABLE_SIZE];
  __shared__ u64 q_shared_results_64[SHARED_Q_RESULTS_SIZE];
  __shared__ u64 tmp_shared_64[SHARED_Q_RESULTS_SIZE];

  int threadsPerBlock = blockDim.x;

  if (threadsPerBlock > SHARED_Q_RESULTS_SIZE)
  {
    printf("we have an error, block dimensions cannot be bigger than shared q results size  ");
    return;
  }

  int totalLoads = TABLE_SIZE / threadsPerBlock; // Each thread loads this many 64 bit values
  int sharedMemoryLoadIndex = threadIdx.x * totalLoads;
  int end = sharedMemoryLoadIndex + totalLoads;

  // Load data into shared memory
  for (int runningLoadIndex = sharedMemoryLoadIndex; runningLoadIndex < end; ++runningLoadIndex)
  {
    // sharedData[runningLoadIndex] = globalData[runningLoadIndex];
    if (runningLoadIndex < TABLE_SIZE)
    { // Ensure we don't go out of bounds of shared memory
      T_shared[runningLoadIndex] = T[runningLoadIndex];
    }
  }

 // Synchronize to ensure all data is loaded
 // __syncthreads();

  int grid_stride_local_index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  u64 *msg_64 = (u64 *)msg;

  u64 *transformation_buffer_q_64 = (u64 *)msg_transformation_buffer;
  int msglen_64 = msglen / COLS512; // length of char* data type to  u64* type

  for (int global_index_64 = grid_stride_local_index; global_index_64 < (msglen_64); global_index_64 += stride)
  {
    if (global_index_64 < msglen_64)
    {
      int q_local_index_shared_64 = global_index_64 % SHARED_Q_RESULTS_SIZE;
      q_shared_results_64[q_local_index_shared_64] = msg_64[global_index_64];

      ROUNDQ512(q_shared_results_64, tmp_shared_64, 0, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(tmp_shared_64, q_shared_results_64, 1, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(q_shared_results_64, tmp_shared_64, 2, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(tmp_shared_64, q_shared_results_64, 3, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(q_shared_results_64, tmp_shared_64, 4, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(tmp_shared_64, q_shared_results_64, 5, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(q_shared_results_64, tmp_shared_64, 6, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(tmp_shared_64, q_shared_results_64, 7, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(q_shared_results_64, tmp_shared_64, 8, (u64 *)T_shared, q_local_index_shared_64);
      ROUNDQ512(tmp_shared_64, q_shared_results_64, 9, (u64 *)T_shared, q_local_index_shared_64);

      // __syncthreads();
      transformation_buffer_q_64[global_index_64] = q_shared_results_64[q_local_index_shared_64];
    }
  }

  
  // Processing P blocks and thread coarsening to 8 threads. P is not very parallelizable 
  if ((blockIdx.x == 0) && (threadIdx.x < 8))
  {
    int tid = threadIdx.x;
    m64_h = (u64 *)outputTransformation;
    output_transformation_shared[tid] = m64_h[tid];
    u64 *current_q_transformed_block = transformation_buffer_q_64;

    int tmpIndex = 0;

    while (msglen >= SIZE512)
    {
      msg_64 = (u64 *)msg;
      m64_hm[tid] = output_transformation_shared[tid] ^ msg_64[tid];

      ROUNDP512(m64_hm, tmp, 0, T_shared, tid);
      ROUNDP512(tmp, m64_hm, 1, T_shared, tid);
      ROUNDP512(m64_hm, tmp, 2, T_shared, tid);
      ROUNDP512(tmp, m64_hm, 3, T_shared, tid);
      ROUNDP512(m64_hm, tmp, 4, T_shared, tid);
      ROUNDP512(tmp, m64_hm, 5, T_shared, tid);
      ROUNDP512(m64_hm, tmp, 6, T_shared, tid);
      ROUNDP512(tmp, m64_hm, 7, T_shared, tid);
      ROUNDP512(m64_hm, tmp, 8, T_shared, tid);
      ROUNDP512(tmp, m64_hm, 9, T_shared, tid);

      output_transformation_shared[tid] = output_transformation_shared[tid] ^ current_q_transformed_block[tid];
      output_transformation_shared[tid] = output_transformation_shared[tid] ^ m64_hm[tid];

      msg += SIZE512;
      current_q_transformed_block += COLS512;
      msglen -= SIZE512;
      tmpIndex++;
    }

//    __syncthreads();

    tmp[tid] = output_transformation_shared[tid];

    ROUNDP512(tmp, tmp_output, 0, T_shared, tid);
    ROUNDP512(tmp_output, tmp, 1, T_shared, tid);
    ROUNDP512(tmp, tmp_output, 2, T_shared, tid);
    ROUNDP512(tmp_output, tmp, 3, T_shared, tid);
    ROUNDP512(tmp, tmp_output, 4, T_shared, tid);
    ROUNDP512(tmp_output, tmp, 5, T_shared, tid);
    ROUNDP512(tmp, tmp_output, 6, T_shared, tid);
    ROUNDP512(tmp_output, tmp, 7, T_shared, tid);
    ROUNDP512(tmp, tmp_output, 8, T_shared, tid);
    ROUNDP512(tmp_output, tmp, 9, T_shared, tid);

    m64_h[tid] = output_transformation_shared[tid] ^ tmp[tid];
    return;
  }
}

/* digest part of a message after identifying variant */
int Transform(u32 *outputTransformation, int outputTransformSize, const u8 *msg, int msglen)
{

  u8 *deviceMessage, *deviceOutputTransform, *deviceTransformationBuffer;
  hipDeviceSynchronize();
  checkCudaErrors(hipMalloc((void **)&deviceMessage, msglen));
  checkCudaErrors(hipMalloc((void **)&deviceTransformationBuffer, msglen)); // to store the q results on device.
  checkCudaErrors(hipMalloc((void **)&deviceOutputTransform, outputTransformSize));
  checkCudaErrors(hipMemcpy(deviceMessage, msg, msglen, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(deviceOutputTransform, outputTransformation, outputTransformSize, hipMemcpyHostToDevice));

  struct hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  dim3 dimBlock(32, 1, 1); // should always be a multiple of 8 so that we can process 8 64bit elements message blocks within cuda blocks. Cannot be greater than 512
  dim3 dimGrid(32, 1, 1);

  getLastCudaError();

  for (int i = 0; i < 1; i++)
  {
    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    Transform512<<<dimGrid, dimBlock>>>((u32 *)deviceOutputTransform, outputTransformSize, (u8 *)deviceMessage, msglen, deviceTransformationBuffer);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %f milliseconds\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceSynchronize();
    getLastCudaError();
  }
  // gettimeofday(&start, NULL);
  checkCudaErrors(hipMemcpy(outputTransformation, deviceOutputTransform, outputTransformSize, hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(deviceTransformationBuffer));
  checkCudaErrors(hipFree(deviceMessage));
  checkCudaErrors(hipFree(deviceOutputTransform));
  hipDeviceSynchronize();
  return 1;
}

/* initialise context */
HashReturn Init(hashState *ctx,
                int hashbitlen)
{
  /* output size (in bits) must be a positive integer less than or
     equal to 512, and divisible by 8 */
  if (hashbitlen <= 0 || (hashbitlen % 8) || hashbitlen > 512)
    return BAD_HASHLEN;

  /* set number of state columns and state size depending on
     variant */
  if (hashbitlen <= 256)
  {
    ctx->columns = COLS512;
    ctx->statesize = SIZE512;
  }
  else
  {
    // ctx->columns = COLS1024;
    // ctx->statesize = SIZE1024;
  }

  /* set other variables */
  ctx->hashbitlen = hashbitlen;
  ctx->block_counter = 0;

  return SUCCESS;
}

/* update state with databitlen bits of input */
HashReturn Update(hashState *ctx,
                  const BitSequence *input,
                  DataLength databitlen, u32 *transformedOutput)
{
  int index = 0;
  const int msglen = (int)(databitlen / 8);
  int newMsgLen = msglen;
  int rem = (int)(databitlen % 8);
  uchar *byteInput = (uchar *)input;

  ctx->block_counter = msglen / ctx->statesize;
  byteInput[newMsgLen] = 0x80;
  newMsgLen++;

  const int remainder = (newMsgLen) % ctx->statesize;
  int remainderIndex = remainder;
  /* store remaining data in buffer */
  if (remainderIndex > ctx->statesize - LENGTHFIELDLEN)
  {
    // extra buffer
    while (remainderIndex < ctx->statesize)
    {
      byteInput[newMsgLen] = 0;
      remainderIndex++;
      newMsgLen++;
    }
    // newMsgLen = newMsgLen + (ctx->statesize - remainder);
    remainderIndex = 0;
    ctx->block_counter++;
  }

  while (remainderIndex < ctx->statesize - LENGTHFIELDLEN)
  {
    byteInput[newMsgLen] = 0;
    remainderIndex++;
    newMsgLen++;
  }
  ctx->block_counter++;

  // byteInput[newMsgLen + (remainderIndex -1 )] = (u8)ctx->block_counter;
  newMsgLen += LENGTHFIELDLEN;

  int lengthPad = LENGTHFIELDLEN;
  int lengthPadIndex = 1;
  while (lengthPadIndex <= LENGTHFIELDLEN)
  {
    byteInput[newMsgLen - lengthPadIndex] = (u8)ctx->block_counter;
    lengthPadIndex++;
    ctx->block_counter >>= 8;
  }

  Transform(transformedOutput, ctx->statesize, input, newMsgLen);
  return SUCCESS;
}

HashReturn Final(hashState *ctx, u32 *input,
                 BitSequence *output)
{
  int i, j = 0, hashbytelen = ctx->hashbitlen / 8;
  u8 *s = (u8 *)input;

  /* store hash result in output */
  for (i = ctx->statesize - hashbytelen; i < ctx->statesize; i++, j++)
  {
    output[j] = s[i];
  }

  /* zeroise relevant variables and deallocate memory */
  for (i = 0; i < ctx->columns; i++)
  {
    input[i] = 0;
  }
  return SUCCESS;
}

/* hash bit sequence */
HashReturn Hash(int hashbitlen,
                const BitSequence *data,
                DataLength databitlen,
                BitSequence *hashval)
{
  HashReturn ret;
  hashState context;

  /* initialise */
  if ((ret = Init(&context, hashbitlen)) != SUCCESS)
    return ret;

  u32 *transformedOutput = (u32 *)calloc(context.statesize, 1);
  /* allocate memory for state and data buffer */
  transformedOutput[2 * context.columns - 1] = U32BIG((u32)context.hashbitlen);

  /* process message */
  if ((ret = Update(&context, data, databitlen, transformedOutput)) != SUCCESS)
    return ret;

  /* finalise */
  ret = Final(&context, transformedOutput, hashval);

  free(transformedOutput);

  return ret;
}

void PrintHash(const BitSequence *hash,
               int hashbitlen)
{
  int i;
  for (i = 0; i < hashbitlen / 8; i++)
  {
    printf("%02x", hash[i]);
  }
  printf("\n");
}

// /* eBash API */
#define crypto_hash_BYTES 32u
#ifdef crypto_hash_BYTES
int crypto_hash(unsigned char *out, const unsigned char *in, u64 inlen)
{
  if (Hash(256, in, inlen * 8, out) == SUCCESS)
    return 0;
  return -1;
}
#endif

void printHexArray(unsigned char *array, uint size)
{
  int i;
  for (i = 0; i < size; i++)
    printf("%02x", array[i]);
  printf("\n");
}

int main(int argc, char **argv)
{
  uint *ct, *pt;
  ct = (uint *)malloc(8 * sizeof(uint)); // Allocating memory for 8 uints

  int dataSize; // Total data size
  size_t maxSharedMemory;
  FILE *file = fopen("text_generator/pt_1MB.txt", "r");

  if (file == NULL)
  {
    printf("Error opening the file.\n");
    return -1;
  }

  fseek(file, 0, SEEK_END);
  dataSize = ftell(file);
  fseek(file, 0, SEEK_SET);

  // Host array
  unsigned char *hostData = (unsigned char *)malloc(dataSize + (SIZE512 * 2));
  if (hostData == NULL)
  {
    printf("Error allocating memory.\n");
    fclose(file);
    return -1;
  }

  fread(hostData, sizeof(unsigned char), dataSize, file);
  fclose(file);

  // const char *message = "my message gdfjhghjkfdhgjklfdshgjklfdhgjkfdshkfjsdhgjfdlshgjkfdsghfjdklhgjfkdlghfjdkslhgfdjksgsdfhj    dsdscxcd3232322cc";
  // size_t size = strlen(message);

  // unsigned char* data = (unsigned char*)malloc(size + (SIZE512 * 2));
  // memcpy(data, message, size);
  // crypto_hash((uchar*)ct, data, size);

  printf("Data: %s\n", hostData);
  printf("Size: %zu\n", dataSize);
  crypto_hash((uchar *)ct, hostData, dataSize);

  printHexArray((uchar *)ct, 32);
  printf("done done\n");
  return 1;
}

#else
#error "MMX instructions must be enabled"
#endif /* __MMX__ */
